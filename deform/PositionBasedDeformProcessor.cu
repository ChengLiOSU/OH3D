#include "hip/hip_runtime.h"
#include "PositionBasedDeformProcessor.h"
#include "Lens.h"
#include "MeshDeformProcessor.h"
#include "Volume.h"
#include "TransformFunc.h"
#include "MatrixManager.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


//!!! NOTE !!! spacing not considered yet!!!! in the global functions


texture<float, 3, hipReadModeElementType>  volumeTexInput;
surface<void, cudaSurfaceType3D>			volumeSurfaceOut;

texture<float, 3, hipReadModeElementType>  channelVolumeTexInput;
surface<void, cudaSurfaceType3D>			channelVolumeSurfaceOut;


__device__ float3 sampleDis(float3 pos, float3 start, float3 end, float r, float deformationScaleVertical, float3 dir2nd)
{
	const float3 noChangeMark = make_float3(-1, -2, -3);
	const float3 emptyMark = make_float3(-3, -2, -1);

	float3 tunnelVec = normalize(end - start);
	float tunnelLength = length(end - start);

	float3 voxelVec = pos - start;
	float l = dot(voxelVec, tunnelVec);
	if (l > 0 && l < tunnelLength){
		float disToStart = length(voxelVec);
		float l2 = dot(voxelVec, dir2nd);
		if (abs(l2) < deformationScaleVertical){
			float3 prjPoint = start + l*tunnelVec + l2*dir2nd;
			float3 dir = normalize(pos - prjPoint);
			float dis = length(pos - prjPoint);
			float3 samplePos = prjPoint + dir*(r - (r - dis) * 2);

			if (dis < r / 2){
				return emptyMark;
			}
			else if (dis < r){
				float3 prjPoint = start + l*tunnelVec + l2*dir2nd;
				float3 dir = normalize(start - prjPoint);
				float3 samplePos = prjPoint + dir*(r - (r - dis) * 2);

				return samplePos;
			}
			else{
				return noChangeMark;
			}
		}
		else{
			return noChangeMark;
		}
	}
	else{
		return noChangeMark;
	}
}



__global__ void
d_updateVolumebyMatrixInfo_rect(hipExtent volumeSize, float3 start, float3 end, float3 spacing, float r, float deformationScaleVertical, float3 dir2nd)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
	{
		return;
	}

	float3 pos = make_float3(x, y, z) * spacing;
	
	float3 tunnelVec = normalize(end - start);
	float tunnelLength = length(end - start);

	float3 voxelVec = pos - start;
	float l = dot(voxelVec, tunnelVec);
	if (l > 0 && l < tunnelLength){
		float disToStart = length(voxelVec);
		float l2 = dot(voxelVec, dir2nd);
		if (abs(l2) < deformationScaleVertical){
			float3 prjPoint = start + l*tunnelVec + l2*dir2nd;
			float3 dir = normalize(pos - prjPoint);
			float dis = length(pos - prjPoint);
			float3 samplePos = prjPoint + dir*(r - (r - dis) * 2);

			if (dis < r / 2){
				float res = 0;
				surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);

				//float res2 = 1;
				//surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
			}
			else if (dis < r){
				float3 prjPoint = start + l*tunnelVec + l2*dir2nd;
				float3 dir = normalize(start - prjPoint);
				float3 samplePos = prjPoint + dir*(r - (r - dis) * 2); //!!! NOTE !!! spacing not considered yet!!!!

				float res = tex3D(volumeTexInput, samplePos.x + 0.5, samplePos.y + 0.5, samplePos.z + 0.5);
				surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);

				//float res2 = tex3D(channelVolumeTexInput, samplePos.x + 0.5, samplePos.y + 0.5, samplePos.z + 0.5);
				//surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
			}
			else{
				float res = tex3D(volumeTexInput, x + 0.5, y + 0.5, z + 0.5);
				surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);

				//float res2 = tex3D(channelVolumeTexInput, x + 0.5, y + 0.5, z + 0.5);
				//surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
			}
		}
		else{
			float res = tex3D(volumeTexInput, x + 0.5, y + 0.5, z + 0.5);
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);

			//float res2 = tex3D(channelVolumeTexInput, x + 0.5, y + 0.5, z + 0.5);
			//surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
		}
	}
	else{
		float res = tex3D(volumeTexInput, x + 0.5, y + 0.5, z + 0.5);
		surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);

		//float res2 = tex3D(channelVolumeTexInput, x + 0.5, y + 0.5, z + 0.5);
		//surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
	}
	return;
}


__global__ void
d_updateVolumebyMatrixInfo_rect_2anime(hipExtent volumeSize, float3 spacing, float3 start, float3 end, float r, float deformationScaleVertical, float3 dir2nd, float lastDeformationDegree, float3 lastDeformationDirVertical, float3 lastTunnelStart, float3 lastTunnelEnd, float rClose)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
	{
		return;
	}

	float3 pos = make_float3(x, y, z) * spacing;

	float3 posOpen = sampleDis(pos, start, end, r, deformationScaleVertical, dir2nd);
	float3 posClose = sampleDis(pos, lastTunnelStart, lastTunnelEnd, rClose, deformationScaleVertical, lastDeformationDirVertical);

	if (posOpen.x < 0 && posOpen.z < posOpen.x && posClose.x < 0 && posClose.z < posClose.x){ //both no change of sample position
		float res = tex3D(volumeTexInput, x + 0.5, y + 0.5, z + 0.5);
		surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
	}
	else if (posOpen.x < 0 && posOpen.z < posOpen.x){//for open no change of sample position, just regular close
		if (posClose.x < 0 && posClose.x < posClose.z){
			float res = 0;
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
		}
		else{
			float res = tex3D(volumeTexInput, posClose.x + 0.5, posClose.y + 0.5, posClose.z + 0.5);
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
		}
	}
	else if (posClose.x < 0 && posClose.z < posClose.x){//for close no change of sample position, just regular open
		if (posOpen.x < 0 && posOpen.x < posOpen.z){
			float res = 0;
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
		}
		else{
			float res = tex3D(volumeTexInput, posOpen.x + 0.5, posOpen.y + 0.5, posOpen.z + 0.5);
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
		}
	}
	else{ //affected by both close and open
		//only work as open
		if (posOpen.x < 0 && posOpen.x < posOpen.z){
			float res = 0;
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
		}
		else{
			float res = tex3D(volumeTexInput, posOpen.x + 0.5, posOpen.y + 0.5, posOpen.z + 0.5);
			surf3Dwrite(res, volumeSurfaceOut, x * sizeof(float), y, z);
		}
	}
	return;
}


__global__ void
d_updateVolumebyMatrixInfo_tunnel_rect(hipExtent volumeSize, float3 start, float3 end, float3 spacing, float r, float deformationScaleVertical, float3 dir2nd)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
	{
		return;
	}

	float3 pos = make_float3(x, y, z) * spacing;
	float3 tunnelVec = normalize(end - start);
	float tunnelLength = length(end - start);

	float3 voxelVec = pos - start;
	float l = dot(voxelVec, tunnelVec);
	if (l > 0 && l < tunnelLength){
		float disToStart = length(voxelVec);
		float l2 = dot(voxelVec, dir2nd);
		if (abs(l2) < deformationScaleVertical){
			float3 prjPoint = start + l*tunnelVec + l2*dir2nd;
			float3 dir = normalize(pos - prjPoint);
			float dis = length(pos - prjPoint);
			float3 samplePos = prjPoint + dir*(r - (r - dis) * 2);

			if (dis < r / 2){
				float res2 = 1;
				surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
			}
			else if (dis < r){
				float3 prjPoint = start + l*tunnelVec + l2*dir2nd;
				float3 dir = normalize(start - prjPoint);
				float3 samplePos = prjPoint + dir*(r - (r - dis) * 2);

				float res2 = tex3D(channelVolumeTexInput, samplePos.x + 0.5, samplePos.y + 0.5, samplePos.z + 0.5);
				surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
			}
			else{
				float res2 = tex3D(channelVolumeTexInput, x + 0.5, y + 0.5, z + 0.5);
				surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
			}
		}
		else{
			float res2 = tex3D(channelVolumeTexInput, x + 0.5, y + 0.5, z + 0.5);
			surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
		}
	}
	else{
		float res2 = tex3D(channelVolumeTexInput, x + 0.5, y + 0.5, z + 0.5);
		surf3Dwrite(res2, channelVolumeSurfaceOut, x * sizeof(float), y, z);
	}
	return;
}

__global__ void
d_posInDeformedChannelVolume(float3 pos, int3 dims, float3 spacing, bool* inChannel)
{
	float3 ind = pos / spacing;
	if (ind.x >= 0 && ind.x < dims.x && ind.y >= 0 && ind.y < dims.y && ind.z >= 0 && ind.z<dims.z) {
		float res = tex3D(channelVolumeTexInput, ind.x + 0.5, ind.y + 0.5, ind.z + 0.5); //?+0.5
		if (res > 0.5)
			*inChannel = true;
		else
			*inChannel = false;
	}
	else{
		*inChannel = true;
	}
}

void PositionBasedDeformProcessor::doDeforme(float degree)
{
	hipExtent size = volume->volumeCuda.size;
	unsigned int dim = 32;
	dim3 blockSize(dim, dim, 1);
	dim3 gridSize(iDivUp(size.width, blockSize.x), iDivUp(size.height, blockSize.y), iDivUp(size.depth, blockSize.z));

	hipChannelFormatDesc cd = volume->volumeCudaOri.channelDesc;
	checkCudaErrors(hipBindTextureToArray(volumeTexInput, volume->volumeCudaOri.content, cd));
	checkCudaErrors(cudaBindSurfaceToArray(volumeSurfaceOut, volume->volumeCuda.content));

	d_updateVolumebyMatrixInfo_rect << <gridSize, blockSize >> >(size, tunnelStart, tunnelEnd, volume->spacing, degree, deformationScaleVertical, rectVerticalDir);
	checkCudaErrors(hipUnbindTexture(volumeTexInput));
	//checkCudaErrors(hipUnbindTexture(channelVolumeTexInput));
}


void PositionBasedDeformProcessor::doDeforme2Tunnel(float degree, float degreeClose)
{
	hipExtent size = volume->volumeCuda.size;
	unsigned int dim = 32;
	dim3 blockSize(dim, dim, 1);
	dim3 gridSize(iDivUp(size.width, blockSize.x), iDivUp(size.height, blockSize.y), iDivUp(size.depth, blockSize.z));

	hipChannelFormatDesc cd = volume->volumeCudaOri.channelDesc;
	checkCudaErrors(hipBindTextureToArray(volumeTexInput, volume->volumeCudaOri.content, cd));
	checkCudaErrors(cudaBindSurfaceToArray(volumeSurfaceOut, volume->volumeCuda.content));

	d_updateVolumebyMatrixInfo_rect_2anime << <gridSize, blockSize >> >(size, volume->spacing, tunnelStart, tunnelEnd, degree, deformationScaleVertical, rectVerticalDir, lastDeformationDegree, lastDeformationDirVertical, lastTunnelStart, lastTunnelEnd, degreeClose);

	checkCudaErrors(hipUnbindTexture(volumeTexInput));
}

void PositionBasedDeformProcessor::doTunnelDeforme(float degree)
{
	hipExtent size = volume->volumeCuda.size;
	unsigned int dim = 32;
	dim3 blockSize(dim, dim, 1);
	dim3 gridSize(iDivUp(size.width, blockSize.x), iDivUp(size.height, blockSize.y), iDivUp(size.depth, blockSize.z));

	hipChannelFormatDesc cd2 = channelVolume->volumeCuda.channelDesc;
	checkCudaErrors(hipBindTextureToArray(channelVolumeTexInput, channelVolume->volumeCudaOri.content, cd2));
	checkCudaErrors(cudaBindSurfaceToArray(channelVolumeSurfaceOut, channelVolume->volumeCuda.content));

	//d_updateVolumebyMatrixInfo_circluar << <gridSize, blockSize >> >(size, tunnelStart, tunnelEnd, volume->spacing, degree);

	d_updateVolumebyMatrixInfo_tunnel_rect << <gridSize, blockSize >> >(size, tunnelStart, tunnelEnd, volume->spacing, deformationScale, deformationScaleVertical, rectVerticalDir);
	checkCudaErrors(hipUnbindTexture(channelVolumeTexInput));
}


void PositionBasedDeformProcessor::computeTunnelInfo()
{
	float3 eyeInLocal = matrixMgr->getEyeInLocal();
	//float3 tunnelAxis = normalize(matrixMgr->recentMove);
	float3 tunnelAxis = normalize(matrixMgr->getViewVecInLocal());

	float3 recentMove = normalize(matrixMgr->recentMove);
	if (dot(recentMove, tunnelAxis) < -0.9){
		tunnelAxis = -tunnelAxis;
	}

	float step = 0.5;
	
	tunnelEnd = eyeInLocal + tunnelAxis*step;
	while (channelVolume->inRange(tunnelEnd) && channelVolume->getVoxel(tunnelEnd) < 0.5){
		tunnelEnd += tunnelAxis*step;
	}
	//tunnelStart = eyeInLocal;
	tunnelStart = eyeInLocal + tunnelAxis*(-2); //better also compute this number (-2)


	//rectVerticalDir = targetUpVecInLocal;
	if (abs(dot(targetUpVecInLocal, tunnelAxis)) < 0.9){
		rectVerticalDir = normalize(cross(cross(tunnelAxis, targetUpVecInLocal), tunnelAxis));
	}
	else{
		rectVerticalDir = matrixMgr->getViewVecInLocal();
	}
	//std::cout << "rectVerticalDir: " << rectVerticalDir.x << " " << rectVerticalDir.y << " " << rectVerticalDir.z << std::endl;
}


bool PositionBasedDeformProcessor::inDeformedCell(float3 pos)
{
	bool* d_inchannel;
	hipMalloc(&d_inchannel, sizeof(bool)* 1);
	hipChannelFormatDesc cd2 = channelVolume->volumeCudaOri.channelDesc;
	checkCudaErrors(hipBindTextureToArray(channelVolumeTexInput, channelVolume->volumeCuda.content, cd2));
	d_posInDeformedChannelVolume << <1, 1 >> >(pos, volume->size, volume->spacing, d_inchannel);
	bool inchannel;
	hipMemcpy(&inchannel, d_inchannel, sizeof(bool)* 1, hipMemcpyDeviceToHost);
	return inchannel;
}


bool PositionBasedDeformProcessor::process(float* modelview, float* projection, int winWidth, int winHeight)
{
	if (!isActive)
		return false;

	float3 eyeInLocal = matrixMgr->getEyeInLocal();

	if (lastVolumeState == ORIGINAL){
		if (volume->inRange(eyeInLocal) && channelVolume->getVoxel(eyeInLocal) < 0.5){
			// in solid area
			// in this case, set the start of deformation
			if (lastEyeState != inWall){
				//eyeFromCellToWall();
				lastVolumeState = DEFORMED;
				lastEyeState = inWall;

				computeTunnelInfo();
				doTunnelDeforme(deformationScale);
				//start a opening animation
				hasOpenAnimeStarted = true;
				hasCloseAnimeStarted = false;
				startOpen = std::clock();
			}
			else if (lastEyeState == inWall){
				//from wall to wall
			}
		}
		else{
			// either eyeInLocal is out of range, or eyeInLocal is in channel
			//in this case, no state change
		}
	}
	else{ //lastVolumeState == Deformed
		if (volume->inRange(eyeInLocal) && channelVolume->getVoxel(eyeInLocal) < 0.5){

			//in area which is solid in the original volume
			bool inchannel = inDeformedCell(eyeInLocal);
			if (inchannel){
				// not in the solid region in the deformed volume
				// in this case, no change
			}
			else{
				//even in the deformed volume, eye is still inside the solid region 
				//eye should just move to a solid region

				//volume->reset();
				//channelVolume->reset();

				lastDeformationDegree = closeStartingRadius;
				lastDeformationDirVertical = rectVerticalDir;
				lastTunnelStart = tunnelStart;
				lastTunnelEnd = tunnelEnd;

				computeTunnelInfo();
				doTunnelDeforme(deformationScale);
	
				hasOpenAnimeStarted = true;//start a opening animation
				hasCloseAnimeStarted = true; //since eye should just moved to the current solid, the previous solid should be closed 
				startOpen = std::clock();
			}
		}
		else{// in area which is channel in the original volume
			hasCloseAnimeStarted = true;
			hasOpenAnimeStarted = false;
			startClose = std::clock();

			channelVolume->reset();
			lastVolumeState = ORIGINAL;
			lastEyeState = inCell;
		}
	}

	if (hasOpenAnimeStarted && hasCloseAnimeStarted){
		float r, rClose;
		double past = (std::clock() - startOpen) / (double)CLOCKS_PER_SEC;
		if (past >= totalDuration){
			r = deformationScale;
			hasOpenAnimeStarted = false;
			hasCloseAnimeStarted = false;
			rClose = 0;
		}
		else{
			r = past / totalDuration*deformationScale;

			if (past >= closeDuration){
				hasCloseAnimeStarted = false;
				rClose = 0;
			}
			else{
				rClose = (1 - past / closeDuration)*closeStartingRadius;
			}

			doDeforme2Tunnel(r, rClose);
		}


	}
	else if (hasOpenAnimeStarted){
		float r;
		double past = (std::clock() - startOpen) / (double)CLOCKS_PER_SEC;
		if (past >= totalDuration){
			r = deformationScale;
			hasOpenAnimeStarted = false;
		}
		else{
			r = past / totalDuration*deformationScale;
			doDeforme(r);
			closeStartingRadius = r;
			closeDuration = past;
		}
	}
	else if (hasCloseAnimeStarted){
		float r;
		double past = (std::clock() - startClose) / (double)CLOCKS_PER_SEC;
		if (past >= closeDuration){
			volume->reset();
			hasCloseAnimeStarted = false;
		}
		else{
			r = (1 - past / closeDuration)*closeStartingRadius;
			doDeforme(r);
		}
	}
	return false;
}


void PositionBasedDeformProcessor::InitCudaSupplies()
{
	volumeTexInput.normalized = false;
	volumeTexInput.filterMode = hipFilterModeLinear;
	volumeTexInput.addressMode[0] = hipAddressModeBorder;
	volumeTexInput.addressMode[1] = hipAddressModeBorder;
	volumeTexInput.addressMode[2] = hipAddressModeBorder;

	channelVolumeTexInput.normalized = false;
	channelVolumeTexInput.filterMode = hipFilterModePoint;
	channelVolumeTexInput.addressMode[0] = hipAddressModeBorder;
	channelVolumeTexInput.addressMode[1] = hipAddressModeBorder;
	channelVolumeTexInput.addressMode[2] = hipAddressModeBorder;
}

