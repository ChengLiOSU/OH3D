#include "hip/hip_runtime.h"
#include <iostream>
#include "ViewpointEvaluator.h"
#include "TransformFunc.h"
#include "Particle.h"
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

texture<float, 3, hipReadModeElementType>  volumeVal;
texture<unsigned short, 3, hipReadModeElementType>  volumeLabel;

texture<float4, 3, hipReadModeElementType>  gradientTexOri;
texture<float4, 3, hipReadModeElementType>  gradientTexFiltered;

ViewpointEvaluator::ViewpointEvaluator(std::shared_ptr<RayCastingParameters> _r, std::shared_ptr<Volume> v)
{
	rcp = _r;

	volume = v;

	volumeVal.normalized = false;
	volumeVal.filterMode = hipFilterModeLinear;
	volumeVal.addressMode[0] = hipAddressModeBorder;
	volumeVal.addressMode[1] = hipAddressModeBorder;
	volumeVal.addressMode[2] = hipAddressModeBorder;

	GPU_setConstants(&(rcp->transFuncP1), &(rcp->transFuncP2), &(rcp->la), &(rcp->ld), &(rcp->ls), &(volume->spacing));
	GPU_setVolume(&(volume->volumeCuda));

	rcp->tstep = 1.0; //generally don't need to sample beyond each voxel

	hipMalloc(&d_hist, sizeof(float)*nbins);

	cubeFaceHists.resize(6);
	for (int i = 0; i < 6; i++){
		hipMalloc(&cubeFaceHists[i], sizeof(float)*nbins);
	}

	cubeInfo.resize(6);
}


void ViewpointEvaluator::initDownSampledResultVolume(int3 sampleSize)
{
	if (resVol != 0)
		resVol.reset();
	resVol = std::make_shared<Volume>();
	resVol->setSize(sampleSize);

	//note that these two rely on the method to set the viewpoint of the sample. also currently most functions do not consider about the origin
	resVol->dataOrigin = indToLocal(0, 0, 0);
	resVol->spacing = indToLocal(1, 1, 1) - resVol->dataOrigin;
}

float3 ViewpointEvaluator::indToLocal(int i, int j, int k)
{
	return make_float3(i - 1, j - 1, k - 1)*make_float3(volume->size.x, volume->size.y, volume->size.z) / make_float3(resVol->size - 3)*volume->spacing;
}

void ViewpointEvaluator::setLabel(std::shared_ptr<VolumeCUDA> v)
{
	volumeLabel.normalized = false;
	volumeLabel.filterMode = hipFilterModePoint;
	volumeLabel.addressMode[0] = hipAddressModeBorder;
	volumeLabel.addressMode[1] = hipAddressModeBorder;
	volumeLabel.addressMode[2] = hipAddressModeBorder;

	checkCudaErrors(hipBindTextureToArray(volumeLabel, v->content, v->channelDesc));

	labelBeenSet = true;
}


void ViewpointEvaluator::initJS06Sphere()
{
	if (JS06SphereInited)	return;

	if (d_r != 0) hipFree(d_r);
	setSpherePoints();
	hipMalloc(&d_r, sizeof(float)*numSphereSample);
	JS06SphereInited = true;
	Tao09DetailInited = false;
}

void ViewpointEvaluator::initTao09Detail()
{
	if (Tao09DetailInited)	return;

	std::cout << "initializing for viewpoint evaluation method" << std::endl;

	if (d_r != 0) hipFree(d_r);
	setSpherePoints();
	hipMalloc(&d_r, sizeof(float)*numSphereSample);
	
	float* gradient = 0;
	volume->computeGradient(gradient);
	volumeGradient.VolumeCUDA_deinit();
	volumeGradient.VolumeCUDA_init(volume->size, gradient, 0, 4);
	delete[] gradient;

	//float* bilateralVolumeRes = 0;
	//volume->computeBilateralFiltering(bilateralVolumeRes, 2, 0.2);
	//FILE * fp = fopen("bilat.raw", "wb");
	//fwrite(bilateralVolumeRes, sizeof(float), volume->size.x*volume->size.y*volume->size.z, fp);
	//fclose(fp);
	
	float* bilateralVolumeRes = new float[volume->size.x*volume->size.y*volume->size.z];
	FILE * fp = fopen("bilat.raw", "rb");
	fread(bilateralVolumeRes, sizeof(float), volume->size.x*volume->size.y*volume->size.z, fp);
	fclose(fp);

	float* bGradient = 0;
	volume->computeGradient(bilateralVolumeRes, volume->size, bGradient);
	filteredVolumeGradient.VolumeCUDA_deinit();
	filteredVolumeGradient.VolumeCUDA_init(volume->size, bGradient, 0, 4);
	delete[] bGradient;
	delete[] bilateralVolumeRes;
		
	gradientTexOri.normalized = false;
	gradientTexOri.filterMode = hipFilterModeLinear;
	gradientTexOri.addressMode[0] = hipAddressModeBorder;
	gradientTexOri.addressMode[1] = hipAddressModeBorder;
	gradientTexOri.addressMode[2] = hipAddressModeBorder;
	
	gradientTexFiltered.normalized = false;
	gradientTexFiltered.filterMode = hipFilterModeLinear;
	gradientTexFiltered.addressMode[0] = hipAddressModeBorder;
	gradientTexFiltered.addressMode[1] = hipAddressModeBorder;
	gradientTexFiltered.addressMode[2] = hipAddressModeBorder;
	
	Tao09DetailInited = true;
	JS06SphereInited = false;
}

void ViewpointEvaluator::compute_UniformSampling(VPMethod m)
{
	float maxRes = -999;
	int3 sampleSize = resVol->size;
	if (m == BS05){
	}
	else if (m == JS06Sphere){
		initJS06Sphere();
		for (int k = 0; k < sampleSize.z; k++){
			std::cout << "now doing k = " << k << std::endl;
			for (int j = 0; j < sampleSize.y; j++){
				for (int i = 0; i < sampleSize.x; i++){
					float3 eyeInLocal = indToLocal(i, j, k);
					float entroRes = computeEntropyJS06Sphere(eyeInLocal);
					resVol->values[k*sampleSize.y*sampleSize.x + j*sampleSize.x + i] = entroRes;
					if (entroRes>maxRes){
						maxRes = entroRes;
						optimalEyeInLocal = eyeInLocal;
					}
				}
			}
		}
	}
	optimalEyeValid = true;
}


void ViewpointEvaluator::compute_SkelSampling(VPMethod m)
{
	float maxRes = -999;
	//int3 sampleSize = resVol->size;
	if (m == BS05){
	}
	else if (m == JS06Sphere){
		initJS06Sphere();
		for (int i = 0; i < skelViews.size(); i++){
			for (int j = 0; j < skelViews[i]->numParticles; j++){
				float3 eyeInLocal = make_float3(skelViews[i]->pos[j]);
				float entroRes = computeEntropyJS06Sphere(eyeInLocal);
				if (entroRes>maxRes){
					maxRes = entroRes;
					optimalEyeInLocal = eyeInLocal;
				}
			}
		}
	}
	else if (m == Tao09Detail){
		initTao09Detail();

		checkCudaErrors(hipBindTextureToArray(gradientTexOri, volumeGradient.content, volumeGradient.channelDesc));
		checkCudaErrors(hipBindTextureToArray(gradientTexFiltered, filteredVolumeGradient.content, filteredVolumeGradient.channelDesc));

		for (int i = 0; i < skelViews.size(); i++){
			for (int j = 0; j < skelViews[i]->numParticles; j++){
				float3 eyeInLocal = make_float3(skelViews[i]->pos[j]);
				float entroRes = computeEntropyTao09Detail(eyeInLocal);
				if (entroRes>maxRes){
					maxRes = entroRes;
					optimalEyeInLocal = eyeInLocal;
				}
			}
		}
		
		checkCudaErrors(hipUnbindTexture(gradientTexOri));
		checkCudaErrors(hipUnbindTexture(gradientTexFiltered));
	}
	optimalEyeValid = true;
}

void ViewpointEvaluator::saveResultVol(const char* fname)
{
	resVol->saveRawToFile(fname);
}

void ViewpointEvaluator::setSpherePoints(int n)
{
	if (spherePointSet) return;

	//source: https://www.openprocessing.org/sketch/41142

	numSphereSample = n;
	sphereSamples.resize(n);

	float phi = (sqrt(5) + 1) / 2 - 1; // golden ratio
	float ga = phi * 2 * M_PI;           // golden angle

	for (int i = 1; i <= numSphereSample; ++i) {
		float lon = ga*i;
		lon /= 2 * M_PI; lon -= floor(lon); lon *= 2 * M_PI;
		if (lon > M_PI)  lon -= 2 * M_PI;

		// Convert dome height (which is proportional to surface area) to latitude
		float lat = asin(-1 + 2 * i / (float)numSphereSample);

		sphereSamples[i - 1] = SpherePoint(lat, lon);
	}
	if (d_sphereSamples != 0){
		hipFree(d_sphereSamples);
	}
	hipMalloc(&d_sphereSamples, sizeof(float)*numSphereSample * 3);
	hipMemcpy(d_sphereSamples, (float*)(&sphereSamples[0]), sizeof(float)*numSphereSample * 3, hipMemcpyHostToDevice);
	spherePointSet = true;
}

__constant__ float colorTableDiverge[33][4] = {
	0, 0.231372549, 0.298039216, 0.752941176,
	0.03125, 0.266666667, 0.352941176, 0.8,
	0.0625, 0.301960784, 0.407843137, 0.843137255,
	0.09375, 0.341176471, 0.458823529, 0.882352941,
	0.125, 0.384313725, 0.509803922, 0.917647059,
	0.15625, 0.423529412, 0.556862745, 0.945098039,
	0.1875, 0.466666667, 0.603921569, 0.968627451,
	0.21875, 0.509803922, 0.647058824, 0.984313725,
	0.25, 0.552941176, 0.690196078, 0.996078431,
	0.28125, 0.596078431, 0.725490196, 1,
	0.3125, 0.639215686, 0.760784314, 1,
	0.34375, 0.682352941, 0.788235294, 0.992156863,
	0.375, 0.721568627, 0.815686275, 0.976470588,
	0.40625, 0.760784314, 0.835294118, 0.956862745,
	0.4375, 0.8, 0.850980392, 0.933333333,
	0.46875, 0.835294118, 0.858823529, 0.901960784,
	0.5, 0.866666667, 0.866666667, 0.866666667,
	0.53125, 0.898039216, 0.847058824, 0.819607843,
	0.5625, 0.925490196, 0.82745098, 0.77254902,
	0.59375, 0.945098039, 0.8, 0.725490196,
	0.625, 0.960784314, 0.768627451, 0.678431373,
	0.65625, 0.968627451, 0.733333333, 0.62745098,
	0.6875, 0.968627451, 0.694117647, 0.580392157,
	0.71875, 0.968627451, 0.650980392, 0.529411765,
	0.75, 0.956862745, 0.603921569, 0.482352941,
	0.78125, 0.945098039, 0.552941176, 0.435294118,
	0.8125, 0.925490196, 0.498039216, 0.388235294,
	0.84375, 0.898039216, 0.439215686, 0.345098039,
	0.875, 0.870588235, 0.376470588, 0.301960784,
	0.90625, 0.835294118, 0.31372549, 0.258823529,
	0.9375, 0.796078431, 0.243137255, 0.219607843,
	0.96875, 0.752941176, 0.156862745, 0.184313725,
	1, 0.705882353, 0.015686275, 0.149019608,
};

__device__ float3 GetColourDiverge2(float v)
{
	//can be accelerated using binary search!!
	int pos = 0;
	bool notFound = true;
	const int numItemColorTableDiverge = 33;
	while (pos < numItemColorTableDiverge - 1 && notFound) {
		if (colorTableDiverge[pos][0] <= v && colorTableDiverge[pos + 1][0] >= v)
			notFound = false;
		pos++;
	}
	float ratio = (v - colorTableDiverge[pos][0]) / (colorTableDiverge[pos + 1][0] - colorTableDiverge[pos][0]);
	
	float3 c = make_float3(
		ratio*(colorTableDiverge[pos + 1][1] - colorTableDiverge[pos][1]) + colorTableDiverge[pos][1],
		ratio*(colorTableDiverge[pos + 1][2] - colorTableDiverge[pos][2]) + colorTableDiverge[pos][2],
		ratio*(colorTableDiverge[pos + 1][3] - colorTableDiverge[pos][3]) + colorTableDiverge[pos][3]);

	return(c);
}


__device__
int intersectBox2(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.d;
	float3 tbot = invR * (boxmin - r.o);
	float3 ttop = invR * (boxmax - r.o);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}



__constant__ float transFuncP1;
__constant__ float transFuncP2;
__constant__ float la;
__constant__ float ld;
__constant__ float ls;
__constant__ float3 spacing;

void ViewpointEvaluator::GPU_setVolume(const VolumeCUDA *vol)
{
	checkCudaErrors(hipBindTextureToArray(volumeVal, vol->content, vol->channelDesc));
}


void ViewpointEvaluator::GPU_setConstants(float* _transFuncP1, float* _transFuncP2, float* _la, float* _ld, float* _ls, float3* _spacing)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(transFuncP1), _transFuncP1, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(transFuncP2), _transFuncP2, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(la), _la, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ld), _ld, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ls), _ls, sizeof(float)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(spacing), _spacing, sizeof(float3)));
}


struct functor_computeEntropy
{
	float sum;
	__device__ __host__ float operator() (float r)
	{
		if (r < 0.00001){
			return 0;
		}
		else{
			float qj = r / sum;
			return -qj*log(qj);
		}
	}
	functor_computeEntropy(float s) : sum(s){}
};





__global__ void d_computeSphereColor(float density, float brightness,
	float3 eyeInLocal, int3 volumeSize, int maxSteps, float tstep, bool useColor, float * r, int numSphereSample, float *sphereSamples, float *hist, int nbins, bool useHist, bool useLabelCount, bool useDist, VPMethod vpmethod)
{

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numSphereSample)	return;

	const float opacityThreshold = 0.95f;


	Ray eyeRay;
	eyeRay.o = eyeInLocal;
	eyeRay.d = make_float3(sphereSamples[3 * i], sphereSamples[3 * i + 1], sphereSamples[3 * i + 2]);

	float tnear, tfar;
	const float3 boxMin = make_float3(0.0f, 0.0f, 0.0f);
	const float3 boxMax = spacing*make_float3(volumeSize); 
	intersectBox2(eyeRay, boxMin, boxMax, &tnear, &tfar);
	tnear = 0.01f;	//!!!NOTE!!! this tnear is not in the clip space but in the original space

	// march along ray from front to back, accumulating color
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float4 sum = make_float4(0.0f); //for JS
	unsigned short label = 0; //for label count
	float detailDescriptor = 0; //for TaoDetail

	float lightingThr = 0.000001; //used for the threshold of TaoDetail

	for (int i = 0; i<maxSteps; i++)
	{
		float3 coord = pos / spacing;
		float sample = tex3D(volumeVal, coord.x, coord.y, coord.z);
		float funcRes = clamp((sample - transFuncP2) / (transFuncP1 - transFuncP2), 0.0, 1.0);

		// lookup in transfer function texture
		float4 col;

		float3 cc;
		if (useColor)
			cc = GetColourDiverge2(clamp(funcRes, 0.0f, 1.0f));
		else
			cc = make_float3(funcRes, funcRes, funcRes);

		////currently ignore lighting
		col = make_float4(la*cc, funcRes);

		col.w *= density;

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		
		float visibility = 1.0f - sum.w;
		
		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		if (vpmethod == Tao09Detail){ //if not Tao09Detail, the sampled texture may not be prepared
			float curDetail = 0;
			float3 normalOri = make_float3(tex3D(gradientTexOri, coord.x, coord.y, coord.z)) / spacing;
			float3 normalFiltered = make_float3(tex3D(gradientTexFiltered, coord.x, coord.y, coord.z)) / spacing;
			if (length(normalOri) > lightingThr){
				if (length(normalFiltered) > lightingThr){
					curDetail = 1 - dot(normalize(normalOri), normalize(normalFiltered));
				}
				else{
					curDetail = 1;
				}
			}
			detailDescriptor = detailDescriptor + curDetail*visibility;
		}

		// exit early if opaque
		if (sum.w > opacityThreshold){
			break;
		}
		else if (useLabelCount){
			unsigned short curlabel = tex3D(volumeLabel, coord.x, coord.y, coord.z);
			if (curlabel > label)
			{
				label = curlabel;
			}
		}

		t += tstep;

		if (t > tfar){
			break;
		}

		pos += step;
	}

	sum *= brightness;

	float uv = sum.w;
	if (vpmethod == Tao09Detail)
		uv = detailDescriptor;

	r[i] = uv;

	if (useLabelCount){
		r[i] = label;
		if (useHist){
			int bin;
			if (label > 0)
				bin = 1;
			else
				bin = 0;
			atomicAdd(hist + bin, 1);
		}
	}
	else if(useDist){
		float dis;
		if (uv < 0.00001)
			dis = 0;
		else
			dis = t;

		r[i] = uv;

		if (useHist){
			float maxDist = fmaxf(fmaxf(boxMax.x, boxMax.y), boxMax.z);
			float minDist = 0;
			// !!! change the range into [0,1] !!!
			int bin = min((int)((dis - minDist) / (maxDist - minDist) *nbins), nbins - 1);
			atomicAdd(hist + bin, 1);		
		}
	}
	else{
		if (useHist){
			// !!! this is true only when we know uv is in [0,1] !!!
			int bin = min((int)(uv*nbins), nbins - 1);
			atomicAdd(hist + bin, 1);
		}
	}
}


struct is_solid
{
	__host__ __device__
	bool operator()(float x)
	{
		return x>0.00001;
	}
};


float ViewpointEvaluator::computeEntropyJS06Sphere(float3 eyeInLocal)
{
	int threadsPerBlock = 64;
	int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

	hipMemset(d_hist, 0, sizeof(float)*nbins);

	if (useLabelCount && !labelBeenSet){
		std::cout << "label not set yet! " << std::endl;
		exit(0);
	}

	d_computeSphereColor << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, d_hist, nbins, useHist, useLabelCount, useDist, JS06Sphere);

	float ret;
	if (useHist){
		if (useLabelCount){
			ret = computeVectorEntropy(d_hist, maxLabel + 1);
		}
		else if (useColor){
			ret = computeVectorEntropy(d_hist, nbins);
		}
	}
	else if (useTrad){
		ret = computeVectorEntropy(d_r, numSphereSample);
	}
	else{
		std::cout << "entropy computation not defined! " << std::endl;
		exit(0);
	}

	return ret;
}

float ViewpointEvaluator::computeEntropyTao09Detail(float3 eyeInLocal)
{
	int threadsPerBlock = 64;
	int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

	hipMemset(d_hist, 0, sizeof(float)*nbins);

	if (useLabelCount && !labelBeenSet){
		std::cout << "label not set yet! " << std::endl;
		exit(0);
	}

	d_computeSphereColor << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, d_hist, nbins, useHist, useLabelCount, useDist, Tao09Detail);

	float ret;
	if (useHist){
		if (useLabelCount){
			ret = computeVectorEntropy(d_hist, maxLabel + 1);
		}
		else if (useColor){
			ret = computeVectorEntropy(d_hist, nbins);
		}
	}
	else if (useTrad){
		ret = computeVectorEntropy(d_r, numSphereSample);
	}
	else{
		std::cout << "entropy computation not defined! " << std::endl;
		exit(0);
	}

	return ret;
}


float ViewpointEvaluator::computeVectorEntropy(float* ary, int size)
{
	thrust::device_vector< float > iVec(ary, ary + size);

	std::vector<float> stl_vector(size);
	thrust::copy(iVec.begin(), iVec.end(), stl_vector.begin());

	float sum = thrust::reduce(iVec.begin(), iVec.end(), (float)0, thrust::plus<float>());
	thrust::transform(iVec.begin(), iVec.end(), iVec.begin(), functor_computeEntropy(sum));
	return thrust::reduce(iVec.begin(), iVec.end(), (float)0, thrust::plus<float>());
}


__global__ void d_computeCubeColorHist(float density, float brightness,
	float3 eyeInLocal, float3 viewVec, float3 upVec, int3 volumeSize, int maxSteps, float tstep, bool useColor, float * r, int numSphereSample, float *sphereSamples, float *hist0, float *hist1, float *hist2, float *hist3, float *hist4, float *hist5, int nbins, bool useHist, bool useLabelCount, VPMethod vpmethod)//, bool useDist)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numSphereSample)	return;

	const float opacityThreshold = 0.95f;

	Ray eyeRay;
	eyeRay.o = eyeInLocal;
	eyeRay.d = make_float3(sphereSamples[3 * i], sphereSamples[3 * i + 1], sphereSamples[3 * i + 2]);

	float tnear, tfar;
	const float3 boxMin = make_float3(0.0f, 0.0f, 0.0f);
	const float3 boxMax = spacing*make_float3(volumeSize);
	intersectBox2(eyeRay, boxMin, boxMax, &tnear, &tfar);
	tnear = 0.01f;	//!!!NOTE!!! this tnear is not in the clip space but in the original space

	// march along ray from front to back, accumulating color
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float4 sum = make_float4(0.0f); //for JS
	unsigned short label = 0; //for label count
	float detailDescriptor= 0; //for TaoDetail

	float lightingThr = 0.000001; //used for the threshold of TaoDetail


	for (int i = 0; i<maxSteps; i++)
	{
		float3 coord = pos / spacing;
		float sample = tex3D(volumeVal, coord.x, coord.y, coord.z);
		float funcRes = clamp((sample - transFuncP2) / (transFuncP1 - transFuncP2), 0.0, 1.0);

		// lookup in transfer function texture
		float4 col;

		float3 cc;
		if (useColor)
			cc = GetColourDiverge2(clamp(funcRes, 0.0f, 1.0f));
		else
			cc = make_float3(funcRes, funcRes, funcRes);

		////currently ignore lighting
		col = make_float4(la*cc, funcRes);

		col.w *= density;

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending

		float visibility = 1.0f - sum.w;

		sum = sum + col*(1.0f - sum.w);


		if (vpmethod == Tao09Detail){ //if not Tao09Detail, the sampled texture may not be prepared
			float curDetail = 0;
			float3 normalOri = make_float3(tex3D(gradientTexOri, coord.x, coord.y, coord.z)) / spacing;
			float3 normalFiltered = make_float3(tex3D(gradientTexFiltered, coord.x, coord.y, coord.z)) / spacing;
			if (length(normalOri) > lightingThr){
				if (length(normalFiltered) > lightingThr){
					curDetail = 1 - dot(normalize(normalOri), normalize(normalFiltered));
				}
				else{
					curDetail = 1;
				}
			}
			detailDescriptor = detailDescriptor + curDetail*visibility;
		}

		// exit early if opaque
		if (sum.w > opacityThreshold){
			break;
		}
		else if (useLabelCount){
			unsigned short curlabel = tex3D(volumeLabel, coord.x, coord.y, coord.z);
			if (curlabel > label)
			{
				label = curlabel;
			}
		}

		t += tstep;
		if (t > tfar){
			break;
		}
		pos += step;
	}
	sum *= brightness;


	int bin;
	if (useLabelCount){
		// !!! this is true only when we know max label is 1 !!!
		if (label > 0)		
			bin = 1;
		else
			bin = 0;
	}
	else{
		if (vpmethod == JS06Sphere){
			float uv = sum.x; // !!! NOTE !!! this is only true for grey scale color
			// !!! this is true only when we know uv is in [0,1] !!!
			bin = min((int)(uv*nbins), nbins - 1);
		}
		else if (vpmethod == Tao09Detail){
			float uv = detailDescriptor;
			// !!! this is true only when we know uv is in [0,1] !!!
			bin = min((int)(uv*nbins), nbins - 1);
		}
	}

	//suppose x coord is along viewVew, suppose upVec and viewVew are normalized and perpendicular
	float3 sidevec = cross(upVec, viewVec);
	float rayz = dot(eyeRay.d, upVec), rayx = dot(eyeRay.d, viewVec), rayy = dot(eyeRay.d, sidevec);

	float xabs = abs(rayx), yabs = abs(rayy), zabs = abs(rayz);
	if (xabs > yabs && xabs > zabs){
		if (rayx > 0){ //front
			atomicAdd(hist0 + bin, 1);
		}
		else{ //back
			atomicAdd(hist1 + bin, 1);
		}
	}
	else if (yabs > xabs && yabs > zabs){
		if (rayy > 0){ //left
			atomicAdd(hist2 + bin, 1);
		}
		else{ //right
			atomicAdd(hist3 + bin, 1);
		}
	}
	else{ //zabs is the max
		if (rayz > 0){ //up
			atomicAdd(hist4 + bin, 1);
		}
		else{ //below
			atomicAdd(hist5 + bin, 1);
		}
	}
}


void ViewpointEvaluator::computeCubeEntropy(float3 eyeInLocal, float3 viewDir, float3 upDir, VPMethod m)
{
	if (m == Tao09Detail){
		initTao09Detail();

		int threadsPerBlock = 64;
		int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

		for (int i = 0; i < 6; i++){
			hipMemset(cubeFaceHists[i], 0, sizeof(float)*nbins);
		}

		if (useLabelCount && !labelBeenSet){
			std::cout << "label not set yet! " << std::endl;
			exit(0);
		}

		checkCudaErrors(hipBindTextureToArray(gradientTexOri, volumeGradient.content, volumeGradient.channelDesc));
		checkCudaErrors(hipBindTextureToArray(gradientTexFiltered, filteredVolumeGradient.content, filteredVolumeGradient.channelDesc));

		d_computeCubeColorHist << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, viewDir, upDir, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, cubeFaceHists[0], cubeFaceHists[1], cubeFaceHists[2], cubeFaceHists[3], cubeFaceHists[4], cubeFaceHists[5], nbins, useHist, useLabelCount, VPMethod::Tao09Detail);

		checkCudaErrors(hipUnbindTexture(gradientTexOri));
		checkCudaErrors(hipUnbindTexture(gradientTexFiltered));

		for (int i = 0; i < 6; i++){
			if (useHist){
				if (useLabelCount){
					cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], maxLabel + 1);
				}
				else if (useColor){
					cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], nbins);
				}
			}
			else{
				std::cout << "entropy computation not defined! " << std::endl;
				exit(0);
			}
		}
	}
	else{
		return;
	}

}

void ViewpointEvaluator::computeCubeEntropy(float3 eyeInLocal, float3 viewDir, float3 upDir)
{
	int threadsPerBlock = 64;
	int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

//	hipMemset(d_hist, 0, sizeof(float)*nbins);
	for (int i = 0; i < 6; i++){
		hipMemset(cubeFaceHists[i], 0, sizeof(float)*nbins);
	}

	if (useLabelCount && !labelBeenSet){
		std::cout << "label not set yet! " << std::endl;
		exit(0);
	}

	d_computeCubeColorHist << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, viewDir, upDir, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, cubeFaceHists[0], cubeFaceHists[1], cubeFaceHists[2], cubeFaceHists[3], cubeFaceHists[4], cubeFaceHists[5], nbins, useHist, useLabelCount, VPMethod::JS06Sphere);

	for (int i = 0; i < 6; i++){
		
		if (useHist){
			if (useLabelCount){
				cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], maxLabel + 1);
			}
			else if (useColor){
				cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], nbins);
			}
		}
		else{
			std::cout << "entropy computation not defined! " << std::endl;
			exit(0);
		}

	}
	return;
}