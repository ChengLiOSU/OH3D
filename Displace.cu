#include "hip/hip_runtime.h"
#include <Displace.h>
#include <TransformFunc.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>
#include <Lens.h>
#include <hip/hip_math_constants.h>

void Displace::LoadOrig(float4* v, int num)
{
	posOrig.assign(v, v + num);// , posOrig.begin());
	d_vec_posScreenTarget.assign(num, make_float2(0, 0));
	d_vec_glyphSizeTarget.assign(num, 1);
	//d_vec_Dist2LensBtm.assign(num, 0);
}

struct functor_Object2Clip//: public thrust::unary_function<float,float>
{
	__device__ __host__ float4 operator() (float4 p)
	{
		return Object2Clip(p, &mv.v[0].x, &pj.v[0].x);
	}
	matrix4x4 mv, pj;
	functor_Object2Clip(matrix4x4 _mv, matrix4x4 _pj) : mv(_mv), pj(_pj){}
};

struct functor_Clip2Screen
{
	int w, h;
	__device__ float2 operator() (float4 p)
	{
		return Clip2ScreenGlobal(GetXY(p), w, h);
	}
	functor_Clip2Screen(int _w, int _h) :w(_w), h(_h){}
};

__device__ __host__ inline float G(float x, float r)
{
	return pow((r - 1), 2) / (-r * r * x + r) + 2 - 1 / r;
}

__device__ __host__ inline float G_Diff(float x, float r)
{
	return pow((r - 1)/ (r * x - 1), 2);
}

__device__ __host__ float2 DisplaceCircleLens(float x, float y, float r, float2 screenPos, float& glyphSize, float focusRatio, float rSide = 0)
{
	float2 ret = screenPos;
	float2 dir = screenPos - make_float2(x, y);
	float disOrig = length(dir);
	float rOut = (r + rSide) / focusRatio; //including the focus and transition region
	if (disOrig < rOut) {
		float disNew = G(disOrig / rOut, focusRatio) * rOut;
		ret = make_float2(x, y) + dir / disOrig * disNew;
		glyphSize = G_Diff(disOrig / rOut, focusRatio);
	}
	return ret;
}

struct functor_Displace
{
	int x, y, r;
	float d;
	float focusRatio;
	float sideSize;
	template<typename Tuple>
	__device__ __host__ void operator() (Tuple t){//float2 screenPos, float4 clipPos) {
		float2 screenPos = thrust::get<0>(t);
		float4 clipPos = thrust::get<1>(t);
		float2 ret = screenPos;
		if (clipPos.z < d) {
			float glyphSize = 1;
			ret = DisplaceCircleLens(x, y, r, screenPos, glyphSize, focusRatio, (d - clipPos.z) * r * 64 * sideSize);
			thrust::get<3>(t) = glyphSize;
		}
		thrust::get<2>(t) = ret;
	}
	functor_Displace(int _x, int _y, int _r, float _d, float _focusRatio, float _sideSize) 
		: x(_x), y(_y), r(_r), d(_d), focusRatio(_focusRatio), sideSize(_sideSize){}
};

struct functor_ApproachTarget
{
	template<typename Tuple>
	__device__ float2 operator() (Tuple t) {
		float2 screenPos = thrust::get<0>(t); 
		float2 screenTarget = thrust::get<1>(t);
		float2 dir = screenTarget - screenPos;
		float sizeDiff = thrust::get<3>(t) - thrust::get<2>(t);
		if (length(dir) < 0.5) {
			thrust::get<0>(t) = screenTarget;
			thrust::get<2>(t) = thrust::get<3>(t);
		}
		else{
			thrust::get<0>(t) = screenPos + dir * 0.1;
			thrust::get<2>(t) = thrust::get<2>(t) + sizeDiff * 0.1;
		}

	}
};

struct functor_Unproject
{
	matrix4x4 inv_mv, inv_pj;
	int w, h;
	__device__ float4 operator() (float4 pClip, float2 pScreen)
	{
		float2 clip = Screen2Clip(pScreen, w, h);
		float4 clip2 = make_float4(clip.x, clip.y, pClip.z, pClip.w);
		return Clip2ObjectGlobal(clip2, &inv_mv.v[0].x, &inv_pj.v[0].x);
	}
	functor_Unproject(matrix4x4 _inv_mv, matrix4x4 _inv_pj, int _w, int _h) :
		inv_mv(_inv_mv), inv_pj(_inv_pj), w(_w), h(_h){}
};

//struct func_CompDist2LensBtm{
//	float3 c;
//	matrix4x4 mv;
//	__device__ float operator() (float4 pObj){
//		float4 a = mat4mulvec4(&(mv.v[0].x), make_float4(c.x, c.y, c.z, 1.0f));
//		float4 b = mat4mulvec4(&(mv.v[0].x), pObj);
//		return abs(a.z / a.w - b.z / b.w);//projection * modelview * v;
//	}
//	func_CompDist2LensBtm(float3 _c, matrix4x4 _mv) : c(_c), mv(_mv){}
//};
//thrust::transform(posOrig.begin(), posOrig.end(), d_vec_Dist2LensBtm.begin(), (l->c, modelview));

void Displace::DisplacePoints(std::vector<float2>& pts, std::vector<Lens*> lenses)
{
	for (int i = 0; i < lenses.size(); i++) {
		CircleLens* l = (CircleLens*)lenses[i];
		for (auto& p : pts) {
			float tmp = 1;
			p = DisplaceCircleLens(l->x, l->y, l->radius, p, tmp, focusRatio);
		}
	}
}

void Displace::Compute(float* modelview, float* projection, int winW, int winH,
	std::vector<Lens*> lenses, float4* ret, float* glyphSizeScale)
{
	if (lenses.size() <= 0)
		return;
	int size = posOrig.size();

	//clip coordiates of streamlines
	matrix4x4 mv(modelview);
	matrix4x4 pj(projection);

	thrust::device_vector<float4> d_vec_posClip(size);
	thrust::device_vector<float2> d_vec_posScreen(size);
	//thrust::counting_iterator < int > first(0);

	if (recomputeTarget) {
		thrust::transform(posOrig.begin(), posOrig.end(), d_vec_posClip.begin(), functor_Object2Clip(mv, pj));

		thrust::transform(d_vec_posClip.begin(), d_vec_posClip.end(),
			d_vec_posScreen.begin(), functor_Clip2Screen(winW, winH));

		//reset to 1
		d_vec_glyphSizeTarget.assign(size, 1);


		for (int i = 0; i < lenses.size(); i++) {
			CircleLens* l = (CircleLens*)lenses[i];
			//thrust::transform(posOrig.begin(), posOrig.end(), 
			//	d_vec_Dist2LensBtm.begin(), func_CompDist2LensBtm(l->c, mv));
			thrust::for_each(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				d_vec_posScreen.begin(), 
				d_vec_posClip.begin(), 
				d_vec_posScreenTarget.begin(),
				d_vec_glyphSizeTarget.begin()
				)),
				thrust::make_zip_iterator(
				thrust::make_tuple(
				d_vec_posScreen.end(),
				d_vec_posClip.end(),
				d_vec_posScreenTarget.end(),
				d_vec_glyphSizeTarget.end()
				)),
				functor_Displace(l->x, l->y, l->radius, l->GetClipDepth(modelview, projection), focusRatio, sideSize));

		}
		

		recomputeTarget = false;
	}



	thrust::device_vector<float4> d_vec_posCur(size);
	thrust::copy(ret, ret + size, d_vec_posCur.begin());
	thrust::device_vector<float> d_vec_glyphSizeScale(size);
	thrust::copy(glyphSizeScale, glyphSizeScale + size, d_vec_glyphSizeScale.begin());

	thrust::transform(d_vec_posCur.begin(), d_vec_posCur.end(), d_vec_posClip.begin(), functor_Object2Clip(mv, pj));
	thrust::transform(d_vec_posClip.begin(), d_vec_posClip.end(),
		d_vec_posScreen.begin(), functor_Clip2Screen(winW, winH));

	thrust::for_each(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		d_vec_posScreen.begin(), 
		d_vec_posScreenTarget.begin(), 
		d_vec_glyphSizeScale.begin(),
		d_vec_glyphSizeTarget.begin()
		)),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		d_vec_posScreen.end(),
		d_vec_posScreenTarget.end(),
		d_vec_glyphSizeScale.end(),
		d_vec_glyphSizeTarget.end()
		)),
		functor_ApproachTarget());

	//posScreenTarget = d_vec_posScreen;
	//}

	matrix4x4 invMV;
	matrix4x4 invPJ;
	invertMatrix(&mv.v[0].x, &invMV.v[0].x);
	invertMatrix(&pj.v[0].x, &invPJ.v[0].x);

	thrust::device_vector<float4> d_vec_ret(size);
	thrust::transform(d_vec_posClip.begin(), d_vec_posClip.end(), d_vec_posScreen.begin(), d_vec_ret.begin(),
		functor_Unproject(invMV, invPJ, winW, winH));
	thrust::copy(d_vec_ret.begin(), d_vec_ret.end(), ret);
	thrust::copy(d_vec_glyphSizeScale.begin(), d_vec_glyphSizeScale.end(), glyphSizeScale);
}
